#include <ATen/ATen.h>
#include <ATen/ExpandUtils.h>
#include <ATen/InitialTensorOptions.h>
#include <ATen/NativeFunctions.h>
#include <ATen/SparseCsrTensorImpl.h>
#include <ATen/SparseCsrTensorUtils.h>
#include <ATen/SparseTensorUtils.h>
#include <ATen/WrapDimUtilsMulti.h>
#include <ATen/native/BinaryOps.h>
#include <ATen/native/Resize.h>
#include <algorithm>

#include <hip/hip_runtime.h>
#include <type_traits>

#include <THC/THCThrustAllocator.cuh>

#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDADataType.h>
#include <ATen/cuda/CUDASparseDescriptors.h>
#include <ATen/cuda/CUDAUtils.h>
#include <c10/cuda/CUDACachingAllocator.h>

#include <ATen/native/sparse/cuda/SparseCUDABlas.cuh>
#include <ATen/native/sparse/cuda/SparseCUDATensorMath.cuh>

#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <thrust/for_each.h>
#include <thrust/sequence.h>

namespace at {
namespace native {

using namespace at::sparse_csr;
// certain utiliy functions are usable from sparse COO.
using namespace at::sparse;

namespace {

void addmm_out_sparse_csr_dense_impl_cuda(const Tensor& input, const SparseCsrTensor& mat1, const Tensor& mat2, const Scalar& beta_, const Scalar& alpha_, const Tensor& result) {

  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND2(kHalf, kBFloat16, result.scalar_type(), "addmm_out_sparse_csr_dense_impl_cuda", [&] {
    auto beta = beta_.to<scalar_t>();
    auto alpha = alpha_.to<scalar_t>();

    if (beta == scalar_t(0)) {
      result.zero_();
    } else if (!is_same_tensor(input, result)) {
      result.copy_(input);
    }

    hipsparseOperation_t opA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipsparseOperation_t opB = HIPSPARSE_OPERATION_NON_TRANSPOSE;

    auto descA = at::cuda::sparse::CuSparseSpMatCsrDescriptor(mat1);
    auto descB = at::cuda::sparse::CuSparseDnMatDescriptor(mat2);
    auto descC = at::cuda::sparse::CuSparseDnMatDescriptor(result);

    hipDataType compute_type = at::cuda::getCudaDataType<scalar_t>();

    auto handle = at::cuda::getCurrentCUDASparseHandle();

    // hipsparseSpMM_bufferSize returns the bufferSize that can be used by hipsparseSpMM
    size_t buffer_size;
    TORCH_CUDASPARSE_CHECK(hipsparseSpMM_bufferSize(
      handle, opA, opB,
      &alpha,
      descA.descriptor(),
      descB.descriptor(),
      &beta,
      descC.descriptor(),
      compute_type,
      HIPSPARSE_SPMM_CSR_ALG2,
      &buffer_size // output
    ));

    auto& allocator = *c10::cuda::CUDACachingAllocator::get();
    auto work_data = allocator.allocate(buffer_size);

    TORCH_CUDASPARSE_CHECK(hipsparseSpMM(
      handle, opA, opB,
      &alpha,
      descA.descriptor(),
      descB.descriptor(),
      &beta,
      descC.descriptor(),
      compute_type,
      HIPSPARSE_SPMM_CSR_ALG2,
      work_data.get()
    ));
  });
}
} // anonymous namespace

Tensor& addmm_out_sparse_csr_dense_cuda(
  const Tensor& self,
  const SparseCsrTensor& sparse,
  const Tensor& dense,
  const Scalar& beta,
  const Scalar& alpha,
  Tensor& r)
{

  TORCH_INTERNAL_ASSERT(sparse.is_sparse_csr());
  Tensor t = *expand_size(self, {sparse.size(0), dense.size(1)}, "addmm_out_sparse_csr");

  TORCH_CHECK(t.is_cuda(),  "Expected all tensors to be on the same device. addmm expected 't' to be CUDA tensor");
  TORCH_CHECK(
      r.is_cuda(),
      "Expected all tensors to be on the same device. addmm: expected 'out' to be CUDA tensor, but got CPU tensor");
  TORCH_CHECK(
      sparse.is_cuda(),
      "Expected all tensors to be on the same device. addmm: expected 'mat1' to be a CUDA tensor, but got a CPU tensor");
  TORCH_CHECK(
      dense.is_cuda(),
      "Expected all tensors to be on the same device. addmm: expected 'mat2' to be a CUDA tensor, but got a CPU tensor");

  TORCH_CHECK(
      sparse.dim() == 2,
      "addmm: 2-D matrices expected, got ",
      sparse.dim(),
      "D tensor");
  TORCH_CHECK(
      dense.dim() == 2,
      "addmm: 2-D matrices expected, got ",
      dense.dim(),
      "D tensor");

  TORCH_CHECK(
      r.is_contiguous() || r.transpose(-2, -1).is_contiguous(),
      "out argument must be contiguous, but got: ",
      r.suggest_memory_format());

  // mxk * kxn = mxn
  int64_t m = sparse.size(0);
  int64_t k = sparse.size(1);
  int64_t n = dense.size(1);

  TORCH_CHECK(
      dense.size(0) == k,
      "addmm: Expected dense matrix (dense) size(0)=",
      k,
      ", got ",
      dense.size(0));

  resize_output(r, {m, n});
  int64_t nnz = sparse._nnz();

  if (nnz == 0) {
    at::mul_out(r, t, at::scalar_tensor(beta, r.options()));
    return r;
  }
  // TODO: Check if hipsparseSpMM can use 64-bit indices
  // https://docs.nvidia.com/cuda/cusparse/index.html
  // auto col_indices = sparse.col_indices().to(at::kInt);
  // auto crow_indices = sparse.crow_indices().to(at::kInt);
  // auto values = sparse.values();

  // s_addmm_out_csr_sparse_dense_cuda_worker(nnz, m, n, k, r, beta, t, alpha, crow_indices, col_indices, values, dense);
  addmm_out_sparse_csr_dense_impl_cuda(self, sparse, dense, beta, alpha, r);
  return r;
}

Tensor& add_out_dense_sparse_csr_cuda(
    Tensor& output,
    const Tensor& dense,
    const SparseCsrTensor& src,
    const Scalar& alpha) {
  TORCH_INTERNAL_ASSERT(dense.layout() == kStrided);
  TORCH_INTERNAL_ASSERT(src.is_sparse_csr());
  TORCH_INTERNAL_ASSERT(dense.is_cuda());

  TORCH_CHECK(
      output.is_contiguous(),
      "out argument must be contiguous, but got: ",
      output.suggest_memory_format());
  TORCH_CHECK(
      output.is_cuda(),
      "add: expected 'out' to be CUDA tensor, but got tensor on device: ",
      output.device());

  TORCH_CHECK(
      src.is_cuda(),
      "add: expected 'other' to be a CUDA tensor, but got tensor on device: ",
      src.device());

  TORCH_CHECK(
      dense.sizes().equals(src.sizes()),
      "add: expected 'self' and 'other' to have same size, but self has size ",
      dense.sizes(),
      " while other has size ",
      src.sizes(),
      " (FYI: dense-sparse addition does not currently support broadcasting)");

  auto commonDtype = promoteTypes(dense.scalar_type(), src.scalar_type());
  TORCH_CHECK(
      canCast(commonDtype, output.scalar_type()),
      "Can't convert result type ",
      commonDtype,
      " to output ",
      output.scalar_type(),
      " in add operation");

  Tensor src_values = src.values();
  Tensor src_crow_indices = src.crow_indices();
  Tensor src_col_indices = src.col_indices();

  resize_output(output, dense.sizes());

  Tensor resultBuffer = output;
  Tensor valuesBuffer = src_values.to(commonDtype);
  if (output.scalar_type() != commonDtype) {
    resultBuffer = dense.to(commonDtype);
  } else if (!is_same_tensor(output, dense)) {
    resultBuffer.copy_(dense);
  }
  AT_DISPATCH_ALL_TYPES(
      commonDtype,
      "add_out_op2_sparse_csr",
      [&valuesBuffer, &resultBuffer, &alpha, &src_crow_indices, &src_col_indices]() {
        AT_DISPATCH_INDEX_TYPES(
            src_crow_indices.scalar_type(),
            "csr_add_out_crow_indices",
              [&valuesBuffer, &resultBuffer, &alpha, &src_crow_indices, &src_col_indices]() {
                scalar_t* values_accessor = valuesBuffer.data_ptr<scalar_t>();
                scalar_t* out_ptr = resultBuffer.data_ptr<scalar_t>();
                scalar_t cast_value = alpha.to<scalar_t>();

                index_t* crow_indices_accessor = src_crow_indices.data_ptr<index_t>();
                index_t* col_indices_accessor = src_col_indices.data_ptr<index_t>();
                int64_t out_storage_offset = resultBuffer.storage_offset();

                auto out_strides = resultBuffer.strides();
                int64_t out_strides0 = out_strides[0];
                int64_t out_strides1 = out_strides[1];

                hipStream_t stream = at::cuda::getCurrentCUDAStream();
                auto allocator = THCThrustAllocator(globalContext().lazyInitCUDA());
                auto policy = thrust::cuda::par(allocator).on(stream);

               // Note that this could be wildly imbalanced if the sparsity pattern varies a lot between rows.
               thrust::for_each(
                    policy,
                    thrust::make_counting_iterator(int64_t(0)),
                    thrust::make_counting_iterator(int64_t(src_crow_indices.size(0) - 1)),
                    [values_accessor,
                    crow_indices_accessor,
                    col_indices_accessor,
                    out_ptr,
                    out_storage_offset,
                    out_strides0,
                    cast_value,
                    out_strides1
                    ]__device__(int64_t irow) {
                        index_t start_index = crow_indices_accessor[irow];
                        index_t end_index = crow_indices_accessor[irow + 1];

                        for (index_t i = start_index; i < end_index; ++i) {
                            auto icol = col_indices_accessor[i];
                            auto index = out_storage_offset + irow * out_strides0 + icol * out_strides1;
                            out_ptr[index] += cast_value * values_accessor[i];
                        }
                    });
              });
      });
  if (output.scalar_type() != commonDtype) {
    output.copy_(resultBuffer);
  }
  return output;
}

Tensor& add_out_sparse_csr_cuda(
    const Tensor& self,
    const SparseCsrTensor& other,
    const Scalar& alpha,
    SparseCsrTensor& out) {
  if (self.layout() == kStrided) {
    return add_out_dense_sparse_csr_cuda(out, self, other, alpha);
  } else {
    TORCH_CHECK(
        false,
        "NotImplementedError: Addition of sparse CSR tensors is not yet implemented.")
  }
  return out;
}

} // namespace native
} // namespace at
